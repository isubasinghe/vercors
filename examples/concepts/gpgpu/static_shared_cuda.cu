//:: cases StaticSharedCuda
//:: tool silicon
//:: verdict Pass

#include <hip/hip_runtime.h>

/*@
  context blockDim.x > 1 && blockDim.y == 1 && blockDim.z == 1;
  context gridDim.x > 0 && gridDim.y == 1 && gridDim.z == 1;

  context in != NULL && out != NULL;
  context \pointer_length(in) == 1;
  context \pointer_length(out) == n;
  context n > 0;
  context blockDim.x * gridDim.x >= n;
  context Perm(&in[0], write \ (blockDim.x * gridDim.x));
  context \gtid<n ==> Perm(&out[\gtid], write);

  requires \ltid == 0 ==> Perm(&s[0], write);

  ensures \gtid<n ==> out[\gtid] == \old(out[\gtid]) + in[0];
@*/
__global__ void blur_x(int* in, int* out, int n) {
  __shared__ int s[1];
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadIdx.x == 0) {
    s[threadIdx.x] = in[0];
  }

  /*@
    context Perm(&in[0], write \ (blockDim.x * gridDim.x));
    context tid<n ==> Perm(&out[tid], write);
    context tid<n ==> \old(out[tid]) == out[tid];

    requires threadIdx.x == 0 ==> Perm(&s[0], write);
    requires threadIdx.x == 0 ==> s[0] == in[0];

    ensures Perm(&s[0], write \ blockDim.x);

    ensures s[0] == in[0];
  @*/
  __syncthreads();

  if(tid < n) {
    out[tid] += s[0];
  }
}